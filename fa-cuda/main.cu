#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "FireflyAlgorithm.h"

#define NUMBER_OF_ITERATIONS 10000

using namespace std;

/*#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}*/

__global__ void add(int *a, int *b, int *c)
{
    printf("done\n");
   *c = *a + *b;
}

int main() {

    std::cout << "Start main" << std::endl;
    int a, b, c; // kopie a, b, c na hoscie
    int *d_a, *d_b, *d_c; // kopie a, b, c na GPU
    int size = sizeof(int);

    // Alokacja pamięci na GPU dla kopii a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // ustawienie wartości początkowych
    a = 5;
    b = 10;

    // Kopiowanie wejść do GPU
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    std::cout << c << std::endl;

    // Wykonanie kernela add() na GPU
    add<<<1,1>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();

    // Kopiowanie rezultatu obliczeń do pamięci hosta
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    // Zwalnianie pamięci
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    std::cout << "End main" << c << std::endl;

    const clock_t begin_time = clock();

    //define parameters
    const int numberOfFireflies = 12;//5;
    const int numberOfDimensions = 20;//3;
    const int dimensionRange = 40;
    const float attractivenessFactor = 1.0f; // attractivityFactor (recommended values: 0.5 - 4)
    const float absorptionFactor = 1e-5; // absorptionFactor (recommended values: 1e-9 - 1e-3)

    FireflyAlgorithm algorithm =  FireflyAlgorithm(numberOfFireflies,numberOfDimensions,
                                                   dimensionRange, attractivenessFactor, absorptionFactor);
    //run task no. 1
    algorithm.RunAlgorithm(NUMBER_OF_ITERATIONS, false, false);

    std::cout << "Task no. 1 execution time: " << float( clock () - begin_time ) /  CLOCKS_PER_SEC << std::endl;

    const clock_t begin_time2 = clock();

    //run task no. 2
    algorithm.RunAlgorithm(NUMBER_OF_ITERATIONS, false, true);
    std::cout << "Task no. 2 execution time: " << float( clock () - begin_time2 ) /  CLOCKS_PER_SEC << std::endl;

    return 0;
}